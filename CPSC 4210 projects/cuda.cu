#include "hip/hip_runtime.h"
/* CPSC 4210 Project 2
	Tony Ho
	cuda.cu
	Implement paralleled matrix-matrix multiplication with CUDA

*/

# include <stdlib.h>
# include <stdio.h>
# include <math.h>
# include <omp.h>
# include <unistd.h>
# include <string.h>
# include <getopt.h>
# include <time.h>

double r8_uniform_01(int *seed);

//Get a matrix element
__device__ float get_e(float* matrix, unsigned long long int row, unsigned long long int col, unsigned long long int e);
//Set values in the matrix
__device__ void set_value(float* matrix, float val, unsigned long long int row, unsigned long long int col, unsigned long long int e);
//Get the next sub-block of a matrix
__device__ float* get_block(float* matrix, unsigned long long int row, unsigned long long int col, unsigned long long int e);

//Kernel function, use threads compute the elements of matrix A and add them together in the end
__global__ void kernel_global(float* A, float* B, float* C, unsigned long long int e);
//Implement MM with GPU's global memory
double* matrix_global(float* A, float* B, float* C, unsigned long long int e, int time_run);
//Kernel function, use threads compute the elements of matrix A by shared memory
__global__ void kernel_shared(float* A, float* B, float* C, unsigned long long int e);
//Implement MM with GPU's shared memory
double* matrix_shared(float* A, float* B, float* C, unsigned long long int e, int time_run);

/* Please modify for GPU Experiments */
/* @@@ Shahadat Hossain (SH) March 12, 2018 */
/******************************************************************************/

int main ( int argc, char *argv[] )

/******************************************************************************/
/*
  Purpose:

   <<< SH:  Skeletal c code for performing dense matrix times matrix. >>>
   <<<      a = b*c where a, b, c are matrices of size n X n          >>>


  Licensing:

    This code is distributed under the GNU LGPL license.

  Modified:

   @@@ Shahadat Hossain (SH) Nov 08, 2019

 */
{
    int id;
    unsigned long long int l;
    unsigned long long int m;
    unsigned long long int n;
    double* temp;
    double avg_dt;	//Average elapse time from 3 runs
    double avg_rate;	//Average rate from 3 runs

  printf ( "\n" );
  printf ( "Dense MXM\n" );
  printf ( "  C/OpenMP version.\n" );
  printf ( "\n" );
  printf ( "  Matrix multiplication tests.\n" );

/*
    @@@ SH Note 1b:

     These values need to be read in from command line. Assume that
     l = m = n.
*/
  //l = 4096;
  //m = 4096;
  //n = 4096;

    char option;
    while((option = getopt(argc, argv, "n:h")) != -1)
    {
        switch(option)
        {
            //When user triggered the flag for assigning, assign the dimensions for matrices
            case 'n':
                e = atoi(optarg);
                break;
            case 'h':
                exit(0);
            default:
                exit(1);
        }
    }

    //Allocate memory space for matricies
    float* A = (float*)malloc(e * e * sizeof(float));
    float* B = (float*)malloc(e * e * sizeof(float));
    float* C = (float*)malloc(e * e * sizeof(float));

    int seed = 123456789;  //Seed value used to generate pseudorandom numbers

    //Generate pseudorandom numbers and assign them to the matrices
    //Assign values for A
    for(i = 0; i < e; i++)
    {
        for(j = 0; j < e; j++)
            A[e * i + j] = (float) (r8_uniform_01(&seed));
    }
    //Assign values for B
    for(i = 0; i < e; i++)
    {
        for(j = 0; j < e; j++)
            A[e * i + j] = (float) (r8_uniform_01(&seed));
    }
    //Assign values for C
    for(i = 0; i < e; i++)
    {
        for(j = 0; j < e; j++)
            A[e * i + j] = (float) (r8_uniform_01(&seed));
    }

	printf ( "\n" );
	printf ( "Cuda Processing\n");
  printf (" Element size = %llu\n", e);
	printf ( "Block size: 16\n");
//=====================================================================================================
	printf( "\n" );
	printf( "CUDA global memory:" );
	for (id = 0; id < 3; id++ )
	{
		temp = matrix_global(A, B, C, e, id);
		avg_dt += temp[0];
		avg_rate += temp[1];
	}
	avg_rate = (double) avg_rate/3;
	avg_dt = (double) avg_dt/3;
	printf("Average Elapsed Time dT: %f\n", avg_dt);
	printf("Average Rate: %f\n", avg_rate);
	avg_rate = avg_dt = 0.0;
	printf( "\n" );
	printf( "------------------------------------------------------------------------------" );
//=====================================================================================================
	printf( "\n" );
	printf( "CUDA shared memory:" );
	for (id = 0; id < 3; id++ )
	{
		temp = matrix_shared(A, B, C, e, id);
		avg_dt += temp[0];
		avg_rate += temp[1];
	}
	avg_rate = (double) avg_rate/3;
	avg_dt = (double) avg_dt/3;
	printf("Average Elapsed Time dT: %f\n", avg_dt);
	printf("Average Rate: %f\n", avg_rate);
	avg_rate = avg_dt = 0.0;
	printf( "\n" );
	printf( "------------------------------------------------------------------------------" );
//=====================================================================================================
/*
  Terminate.
*/
    //Deallocating
    free(A);
    free(B);
    free(C);

  printf ( "\n" );
  printf ( "Dense MXM:\n" );
  printf ( "  Normal end of execution.\n" );

  return 0;
}

//Get a matrix element
__device__ float get_e(float* matrix, unsigned long long int row, unsigned long long int col, unsigned long long int e)
{
    return matrix[e * row + col];
}

//Set values in matrix
__device__ void set_value(float* matrix, float value, unsigned long long int row, unsigned long long int col, unsigned long long int e)
{
    matrix[e * row + col] = value;
}
//Get the next block of a matrix
__device__ float* get_block(float* matrix, unsigned long long int row, unsigned long long int col, unsigned long long int e)
{
    float* block;
    block = &matrix[e * 16 * row + 16 * col];  //Block size = 16
    return block;
}
//+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++
//Kernel function, use threads compute the elements of matrix A and add them together in the end
__global__ void kernel_global(float* A, float* B, float* C, unsigned long long int e)
{
    //Acumulated results of a
    float total = 0.0;
    //Go through row and column of thread block
    unsigned long long int row = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned long long int col = threadIdx.x + blockIdx.x * blockDim.x;
    //Stop if out of range
    if( row > e || col > e )
        return;
    //Find the correct element in B'row and C's, store in total, then assign to A
    int i;
    for(i = 0; i < e; i++)
        total += (B[(row * e) + i]) * (C[col + (e * i)]);
    A[(row * e) + col] = total;
}

//Implement MM with GPU's global memory
double* matrix_global(float* A, float* B, float* C, unsigned long long int e, int time_run)
{
    //Arrays to store matrix A, B, C
    float* matrix_a;
    float* matrix_b;
    float* matrix_c;

    float time_elapsed;
    double rate;
    double dt_rate[2];
    unsigned long long int ops;
    //unsigned long long l = e;
    //unsigned long long m = e;
    //unsigned long long n = e;
    int block_size = 16;
    //Size device memory
    unsigned long long int size = e * e * sizeof(float);

    //Allocation A,B,C matrices
    hipMalloc(&matrix_a, size);
    hipMalloc(&matrix_b, size);
    hipMalloc(&matrix_c, size);
    //Copying memory
    hipMemcpy(matrix_a, A, size, hipMemcpyHostToDevice);
    hipMemcpy(matrix_b, B, size, hipMemcpyHostToDevice);
    hipMemcpy(matrix_c, C, size, hipMemcpyHostToDevice);

    //Setup the grid and block dimensions
    dim3 block(block_size, block_size); //16X16 block
    //Setup grid based on block
    dim3 grid((e + block.x - 1) / block.x, (e + block.y - 1) / block.y);

    //Timer
    hipEvent_t start, end;
    //Create CUDA event
    hipEventCreate(&start);
    hipEventCreate(&end);
    //Timer for CUDA start events
    hipEventRecord(start, 0);

    //Run the kernel function on GPU
    kernel_global<<<grid, block>>>(matrix_a, matrix_b, matrix_c, e);
    //Synchronize threads
    hipDeviceSynchronize();

    //Timer for CUDA stop events
    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    //Store the elapsed time
    hipEventElapsedTime(&time_elapsed, start, end);
    //Deallocate the events when done
    hipEventDestroy(start);
    hipEventDestroy(end);
/*
  Generate Report.

*/
    ops = e * e * (e * 2);
    time_elapsed = time_elapsed / 1000.0;
    rate = (double) (ops) / time_elapsed / 1000000.0;

    printf ( "\n" );
    printf ( "CUDA global memory matrix multiplication timing.\n" );
    printf ( "Time Run: %d\n", time_run + 1 );
    printf ( "A(LxN) = B(LxM) * C(MxN)\n" );
    printf ( "L = %llu\n", e );
    printf ( "M = %llu\n", e );
    printf ( "N = %llu\n", e );
    printf ( "Floating point OPS roughly %llu\n", ops );
    printf ( "Elapsed time dT = %f\n", time_elapsed );
    printf ( "Rate = MegaOPS/dT = %f\n", rate );

    //Copy matrix A from device memory to host memory
    hipMemcpy(A, matrix_a, size, hipMemcpyDeviceToHost);

    dt_rate[0] = time_elapsed;
    dt_rate[1] = rate;

    //Deallocate memory space
    hipFree(matrix_a);
    hipFree(matrix_b);
    hipFree(matrix_c);

    return dt_rate;
}

//+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++
//Kernel function, compute the elements of matrix A and add them together on shared memory
__global__ void kernel_shared(float* A, float* B, float* C, unsigned long long int e)
{
    int i, j;
    //Accumulated results of a
    float total = 0.0;
    //Block's row and column
    unsigned long long int block_row = blockIdx.y;
    unsigned long long int block_col = blockIdx.x;
    //Get next block to compute
    float* block_sub_A = get_block(A, block_row, block_col, e);
    //Thread row and col
    unsigned long long int thread_row = threadIdx.y;
    unsigned long long int thread_col = threadIdx.x;
    //Multiply each pairs of the matrices together, the store in total
    for( i = 0; i < e; i++ )
    {
        //Get block from B
        float* block_B = get_block(B, block_row, i, e);
        //Get block from C
        float* block_C = get_block(C, i, block_col, e);

        //Shared memory used to store the tiles of B and C
        //Block size is 16
        __shared__ float shared_B[16][16];
        __shared__ float shared_C[16][16];

        //Load tiles from device memory to shared memory
        shared_B[thread_row][thread_col] = get_e(B, thread_row, thread_col, e);
        shared_C[thread_row][thread_col] = get_e(C, thread_row, thread_col, e);

        //Synchronize threads before computation
        __syncthreads();

        //Multiply the blocks from B and C, then add
        //Block size is 16
        for(j = 0; j < 16; j++)
            total += shared_B[thread_row][j] * shared_C[j][thread_col];

        //Synchronize threads before stop
        //Get ready for the next turn of computation
        __syncthreads();

        //Put the value to A's block
        set_value(block_sub_A, total, thread_row, thread_col, e);
    }
}

//Implement MM with GPU's shared memory
double* matrix_shared(float* A, float* B, float* C, unsigned long long int e, int time_run)
{
    //Arrays to store matrix A, B, C
    float* matrix_a;
    float* matrix_b;
    float* matrix_c;

    float time_elapsed;
    double rate;
    double dt_rate[2];
    unsigned long long int ops;
    //unsigned long long l = e;
    //unsigned long long m = e;
    //unsigned long long n = e;
    int block_size = 16;

    //Size device memory
    unsigned long long int size = e * e * sizeof(float);

    //Allocation A,B,C
    hipMalloc(&matrix_a, size);
    hipMalloc(&matrix_b, size);
    hipMalloc(&matrix_c, size);
    //Copying memory
    hipMemcpy(matrix_a, A, size, hipMemcpyHostToDevice);
    hipMemcpy(matrix_b, B, size, hipMemcpyHostToDevice);
    hipMemcpy(matrix_c, C, size, hipMemcpyHostToDevice);

    //Setup the grid and block dimensions
    dim3 block(block_size, block_size); //16X16 block
    //Setup grid based on block
    dim3 grid((e + block.x - 1) / block.x, (e + block.y - 1) / block.y);

    //Timer
    hipEvent_t start, end;
    //Create CUDA event
    hipEventCreate(&start);
    hipEventCreate(&end);
    //Timer for CUDA start events
    hipEventRecord(start, 0);

    //Run the kernel function on GPU
    kernel_shared<<<grid, block>>>(matrix_a, matrix_b, matrix_c, e);
    //Synchronize threads
    hipDeviceSynchronize();

    //Timer for CUDA stop events
    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    //Store the elapsed time
    hipEventElapsedTime(&time_elapsed, start, end);
    //Deallocate the events when done
    hipEventDestroy(start);
    hipEventDestroy(end);
/*
  Generate Report.

*/
    ops = e * e * (e * 2);
    time_elapsed = time_elapsed / 1000.0;
    rate = (double) (ops) / time_elapsed / 1000000.0;

    printf ( "\n" );
    printf ( "CUDA shared memory matrix multiplication timing.\n" );
    printf ( "Time Run: %d\n", time_run + 1 );
    printf ( "A(LxN) = B(LxM) * C(MxN)\n" );
    printf ( "L = %llu\n", e );
    printf ( "M = %llu\n", e );
    printf ( "N = %llu\n", e );
    printf ( "Floating point OPS roughly %llu\n", ops );
    printf ( "Elapsed time dT = %f\n", time_elapsed );
    printf ( "Rate = MegaOPS/dT = %f\n", rate );

    //Copy matrix A from device memory to host memory
    hipMemcpy(A, matrix_a, size, hipMemcpyDeviceToHost);

    dt_rate[0] = time_elapsed;
    dt_rate[1] = rate;

    //Deallocate memory space
    hipFree(matrix_a);
    hipFree(matrix_b);
    hipFree(matrix_c);

    return dt_rate;
}

/******************************************************************************/

double r8_uniform_01 ( int *seed )

/******************************************************************************/
/*
  Purpose:

    R8_UNIFORM_01 is a unit pseudorandom double precision real number R8.

  Discussion:

    This routine implements the recursion

      seed = 16807 * seed mod ( 2**31 - 1 )
      unif = seed / ( 2**31 - 1 )

    The integer arithmetic never requires more than 32 bits,
    including a sign bit.

  Licensing:

    This code is distributed under the GNU LGPL license.

  Modified:

    11 August 2004

  Author:

    John Burkardt

  Reference:

    Paul Bratley, Bennett Fox, Linus Schrage,
    A Guide to Simulation,
    Springer Verlag, pages 201-202, 1983.

    Bennett Fox,
    Algorithm 647:
    Implementation and Relative Efficiency of Quasirandom
    Sequence Generators,
    ACM Transactions on Mathematical Software,
    Volume 12, Number 4, pages 362-376, 1986.

  Parameters:

    Input/output, int *SEED, a seed for the random number generator.

    Output, double R8_UNIFORM_01, a new pseudorandom variate, strictly between
    0 and 1.
*/
{
  int k;
  double r;

  k = *seed / 127773;

  *seed = 16807 * ( *seed - k * 127773 ) - k * 2836;

  if ( *seed < 0 )
  {
    *seed = *seed + 2147483647;
  }

  r = ( double ) ( *seed ) * 4.656612875E-10;

  return r;
}
